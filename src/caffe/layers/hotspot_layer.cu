#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"
#include "device_atomic_functions.hpp"

#include "caffe/util/math_functions.hpp"
#include "caffe/layers/hotspot_layer.hpp"
#define CV_PI 3.1415926535897932384626433832795
#define GAUSSIAN(x0,y0,x,y) 0.5 / gaussian_std / gaussian_std / CV_PI * exp(-0.5 * (((x0)-(x)) * ((x0)-(x)) + ((y0)-(y)) * ((y0)-(y))) / gaussian_std / gaussian_std)

namespace caffe {

  __device__ __constant__  float kEps= 1e-4;

  template <typename Dtype>
  __global__ void HotspotFoward(const int num, const int num_point, const Dtype gaussian_std,
                                const int data_height, const int data_width, const bool mean_removed,
                                  const int target_height, const int target_width,
                                  const Dtype*  point_data, Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, num * target_height * target_width * num_point) {
      int n = index / (target_height * target_width * num_point);
      int sp = index % (target_height * target_width * num_point);
      int h = sp / (target_width * num_point);
      int pw = sp % (target_width * num_point);
      int w = pw / num_point;
      int p = pw % num_point;
      Dtype p1 = (point_data[n * num_point * 2 + p * 2] / data_width + (mean_removed ? 0.5 : 0))  * target_width;
      Dtype p2 = (point_data[n * num_point * 2 + p * 2 + 1] / data_height + (mean_removed ? 0.5 : 0)) * target_height;
      Dtype temp = GAUSSIAN(p1, p2, w, h);
      if (temp > kEps) {
        top_data[(((n * num_point + p) * target_height + h) * target_width + w)] = temp;
      }
      else {
        top_data[(((n * num_point + p) * target_height + h) * target_width + w)] = 0;
      }
    }
  }

template <typename Dtype>
void HotspotLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* point_data = bottom[0]->gpu_data();
  const int num_point = bottom[0]->shape(1) / 2;
  const int num = bottom[0]->num();

  HotspotFoward<Dtype> << <CAFFE_GET_BLOCKS(num * num_point * height_ * width_),
  CAFFE_CUDA_NUM_THREADS >> >(num, num_point, gaussian_std_,
    data_height_, data_width_, mean_removed_,
    height_, width_,
    point_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void HotspotLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  NOT_IMPLEMENTED;
}


INSTANTIATE_LAYER_GPU_FUNCS(HotspotLayer);


}  // namespace caffe
