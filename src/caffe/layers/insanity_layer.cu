#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/custom_layers.hpp"

namespace caffe {

// CUDA kernele for forward
template <typename Dtype>
__global__ void InsanityForwardTrain(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype* slope_data) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] / slope_data[index];
  }
}

// CUDA kernele for forward
template <typename Dtype>
__global__ void InsanityForwardTest(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype slope_data) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] / slope_data;
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void InsanityBackward(const int n, const int channels, const int dim,
    const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
    const Dtype* slope_data) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) / slope_data[index]);
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void InsanityBackwardTest(const int n, const int channels, const int dim,
                                 const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
                                 const Dtype slope_data) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
                                        + (in_data[index] <= 0) / slope_data);
  }
}

template <typename Dtype>
void InsanityLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();

  // For in-place computation
  if (top[0] == bottom[0] && lb_ < 0) {
    caffe_copy(count, bottom_data, bottom_memory_.mutable_gpu_data());
  }

  if (this->phase_ == TRAIN) {
    Dtype* slope_data =
        static_cast<Dtype*>(alpha.mutable_gpu_data());
	caffe_gpu_rng_uniform<Dtype>(count, lb_, ub_, slope_data);
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
	InsanityForwardTrain<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, channels, dim, bottom_data, top_data, slope_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    InsanityForwardTest<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
		count, channels, dim, bottom_data, top_data, mean_slope);
    CUDA_POST_KERNEL_CHECK;
  }
}

template <typename Dtype>
void InsanityLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();

  // For in-place computation
  if (top[0] == bottom[0] && lb_ < 0) {
    bottom_data = bottom_memory_.gpu_data();
  }

  // Propagate to bottom
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* slope_data = alpha.gpu_data();

    if (this->phase_ == TRAIN) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      InsanityBackward<Dtype> << <CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS >> >(
        count, channels, dim, top_diff, bottom_data, bottom_diff, slope_data);
      CUDA_POST_KERNEL_CHECK;
    } else {
      InsanityBackwardTest<Dtype> << <CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS >> >(
        count, channels, dim, top_diff, bottom_data, bottom_diff, mean_slope);
      CUDA_POST_KERNEL_CHECK;
    }
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(InsanityLayer);


}  // namespace caffe
