#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/custom_layers.hpp"

namespace caffe {

template <typename Dtype>
void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  Dtype normsqr;
  int n = bottom[0]->num();
  int d = bottom[0]->count() / n;
  caffe_gpu_powx(n*d, bottom_data, Dtype(2), squared_data);
  for (int i=0; i<n; ++i) {
    caffe_gpu_asum<Dtype>(d, squared_data+i*d, &normsqr);
    caffe_gpu_scale<Dtype>(d, pow(normsqr, -0.5), bottom_data+i*d, top_data+i*d);
  }
}

template <typename Dtype>
void NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int n = top[0]->num();
  int d = top[0]->count() / n;
  Dtype a;
  for (int i=0; i<n; ++i) {
    caffe_gpu_dot(d, top_data+i*d, top_diff+i*d, &a);
    caffe_gpu_scale(d, a, top_data+i*d, bottom_diff+i*d);
    caffe_gpu_sub(d, top_diff+i*d, bottom_diff+i*d, bottom_diff+i*d);
    caffe_gpu_dot(d, bottom_data+i*d, bottom_data+i*d, &a);
    caffe_gpu_scale(d, Dtype(pow(a, -0.5)), bottom_diff+i*d, bottom_diff+i*d);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(NormalizeLayer);


}  // namespace caffe