#include "hip/hip_runtime.h"
/*
* triplet_loss_layer.cu
*
*/

#include <algorithm>
#include <vector>


#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/custom_layers.hpp"

namespace caffe {

  template <typename Dtype>
  void TripletLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    const int count = bottom[0]->count();
    caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // p
      diff_ap_.mutable_gpu_data());  // a_i-p_i
    caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[2]->gpu_data(),  // n
      diff_an_.mutable_gpu_data());  // a_i-n_i
    caffe_gpu_sub(
      count,
      bottom[1]->gpu_data(),  // p
      bottom[2]->gpu_data(),  // n
      diff_pn_.mutable_gpu_data());  // p_i-n_i

    caffe_gpu_powx(
      count,
      diff_ap_.mutable_gpu_data(),  // a_i-p_i
      Dtype(2),
      diff_sq_ap_.mutable_gpu_data());  // (a_i-p_i)^2
    caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),                                         //alpha
      diff_sq_ap_.gpu_data(),  // (a_i-p_i)^2                // A
      summer_vec_.gpu_data(),                             // x
      Dtype(0.0),                                         //belta
      dist_sq_ap_.mutable_gpu_data());  // \Sum (a_i-p_i)^2  //y

    caffe_gpu_powx(
      count,
      diff_an_.mutable_gpu_data(),  // a_i-n_i
      Dtype(2),
      diff_sq_an_.mutable_gpu_data());  // (a_i-n_i)^2
    caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),                                         //alpha
      diff_sq_an_.gpu_data(),  // (a_i-n_i)^2                // A
      summer_vec_.gpu_data(),                             // x
      Dtype(0.0),                                         //belta
      dist_sq_an_.mutable_gpu_data());  // \Sum (a_i-n_i)^2  //y

    Dtype margin = this->layer_param_.triplet_loss_param().margin();
    Dtype loss(0.0);

    //const Dtype* sampleW = bottom[3]->cpu_data();
    const Dtype sampleW = Dtype(1.0);

    for (int i = 0; i < bottom[0]->num(); ++i) {
      loss += sampleW*std::max(margin + dist_sq_ap_.cpu_data()[i] - dist_sq_an_.cpu_data()[i], Dtype(0.0));
    }
    loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
    top[0]->mutable_cpu_data()[0] = loss;
  }

  template <typename Dtype>
  __global__ void CLLBackward(const int count, const int channels,
                              const Dtype margin, const Dtype alpha,
                              const Dtype* diff, const Dtype* dist_sq_ap_, const Dtype* dist_sq_an_,
                              Dtype *bottom_diff) {
    CUDA_KERNEL_LOOP(i, count) {
      int n = i / channels;  // the num index, to access dist_sq_ap_ and dist_sq_an_
      Dtype mdist(0.0);
      mdist = margin + dist_sq_ap_[n] - dist_sq_an_[n];
      if (mdist > 0.0) {
        //bottom_diff[i] = alpha*sampleW*diff[i];
        bottom_diff[i] = alpha*diff[i];
      }
      else {
        bottom_diff[i] = 0;
      }
    }
  }

  template <typename Dtype>
  void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                             const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    Dtype margin = this->layer_param_.triplet_loss_param().margin();
    const int count = bottom[0]->count();
    const int channels = bottom[0]->channels();

    for (int i = 0; i < 3; ++i) {
      if (propagate_down[i]) {
        const Dtype sign = (i < 2) ? -1 : 1;
        const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
        if (i == 0) {
          // NOLINT_NEXT_LINE(whitespace/operators)
          CLLBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
            count, channels, margin, alpha,
            diff_pn_.gpu_data(),  // the cached eltwise difference between p and n
            dist_sq_ap_.gpu_data(),  // the cached square distance between a and p
            dist_sq_an_.gpu_data(),  // the cached square distance between a and n
            bottom[i]->mutable_gpu_diff());
          CUDA_POST_KERNEL_CHECK;
        }
        else if (i == 1) {
          // NOLINT_NEXT_LINE(whitespace/operators)
          CLLBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
            count, channels, margin, alpha,
            diff_ap_.gpu_data(),  // the cached eltwise difference between a and p
            dist_sq_ap_.gpu_data(),  // the cached square distance between a and p
            dist_sq_an_.gpu_data(),  // the cached square distance between a and n
            bottom[i]->mutable_gpu_diff());

          CUDA_POST_KERNEL_CHECK;
        }
        else if (i == 2) {
          // NOLINT_NEXT_LINE(whitespace/operators)
          CLLBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
            count, channels, margin, alpha,
            diff_an_.gpu_data(),  // the cached eltwise difference between a and n
            dist_sq_ap_.gpu_data(),  // the cached square distance between a and p
            dist_sq_an_.gpu_data(),  // the cached square distance between a and n
            bottom[i]->mutable_gpu_diff());
          CUDA_POST_KERNEL_CHECK;

        }
      }
    }
  }

  INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer);

}  // namespace caffe
